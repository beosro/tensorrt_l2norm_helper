#include "hip/hip_runtime.h"
#include "l2norm_helper.h"


__global__ void sqrtKernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = sqrtf(x[i]);
    }
}

__global__ void rsqrtKernel(
    const int n,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = rsqrtf(x[i]);
    }
}

__global__ void maxKernel(
    const int n,
    const float eps,
    const float* x,
    float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = fmaxf(x[i], eps);
    }
}

bool executeInference(
    hipStream_t stream,
    const int op_type,
    const float eps,
    const int batch_size,
    const int C,
    const int H,
    const int W,
    const void* inputData,
    void* outputData)
{
    const int length = C * H * W;
    float* input = (float*) const_cast<void*>(inputData);
    float* output = (float*) outputData;
    for (int n = 0; n < batch_size; ++n)
    {
        switch(op_type)
        {
          case operation_t::OP_TYPE_MAX:
            maxKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, eps, input, output);
            break;
          case operation_t::OP_TYPE_RSQRT:
            rsqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          case operation_t::OP_TYPE_SQRT:
            sqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          default:
            return 1;
        }
        // Move cursors
        input += length;
        output += length;
    }
    return 0;
}
